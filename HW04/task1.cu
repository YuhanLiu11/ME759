#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "matmul.cuh"

int main(int argc, char** argv) {
    using namespace std;

    long int n = atol(argv[1]);
    int threads_per_block = atoi(argv[2]);
    float *A;
    float *B;
    float *C;
    hipMallocManaged(&A, (n * n) * sizeof(float));
    hipMallocManaged(&B, (n * n) * sizeof(float));
    hipMallocManaged(&C, (n * n) * sizeof(float));
    for (int i = 0; i < n * n; i++) {
        A[i] = 1;
        B[i] = 1;
        // initialize C
        C[i] = 0;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matmul(A, B, C, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << C[n * n - 1] << endl;
    cout << milliseconds << endl;
    hipFree(A);
    hipFree(B);
    hipFree(C);

}