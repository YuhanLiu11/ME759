#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "matmul.cuh"

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n*n) return;
    int i = index / n;
    int j = index % n;
    float Pvalue = 0;
    for (size_t k = 0; k < n; k++) {
        Pvalue += A[i * n + k] * B[k * n + j];
    }
    C[i * n + j] = Pvalue;
}


void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {
    memset(C, 0, (n * n) * sizeof(float));

    int blockNum = (n * n + threads_per_block - 1) / threads_per_block;

    matmul_kernel <<<blockNum, threads_per_block>>> (A, B, C, n);
    hipDeviceSynchronize();
}