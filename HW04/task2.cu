#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "stencil.cuh"

int main(int argc, char** argv) {
    using namespace std;

    long n = stol(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);
    float *image;
    float *mask;
    float *output;
    hipMallocManaged(&image, n * sizeof(float));
    hipMallocManaged(&mask, (2 * R + 1) * sizeof(float));
    hipMallocManaged(&output, n * sizeof(float));

    for (int i= 0; i < n; i++){
        image[i] = 1;
        output[i] = 0;
    }
    for (int i= 0; i < (2*R + 1); i++){
            mask[i] = 1;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    stencil(image, mask, output, n, R, threads_per_block);

    cout << output[n - 1] << endl;
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << milliseconds << endl;
    hipFree(image);
    hipFree(mask);
    hipFree(output);
}