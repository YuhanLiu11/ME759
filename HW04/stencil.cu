#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "stencil.cuh"

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    using namespace std;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s[];
    float *shared_mask = s;
    float *shared_image = (float *)&shared_mask[(2*R+1)];
    float *shared_results = (float *)&shared_image[blockDim.x + (2*R)];
    if (threadIdx.x == 0) {
        for (int i = 0; i < (2 * R + 1); i++) {
            shared_mask[i] = mask[i];
        }
        for (int i = 0; i < blockDim.x; i++) {
            shared_results[i] = 0;
        }


    }


    if (threadIdx.x == 0) {
        for (int i = 0; i < R+1; i ++) {
            int image_index = index - R + i;
            if (image_index < 0 || image_index >= n) {
                shared_image[i] = 0;
            } else {
                shared_image[i] = image[index - R + i ];
            }
        }

    } else if (threadIdx.x == blockDim.x - 1) {

        for (int i = 0; i < R+1; i ++) {
            int image_index = index + i;

            if (image_index < 0 || image_index >= n) {
                shared_image[i + R + blockDim.x - 1] = 0;

            } else {
                shared_image[i + R + blockDim.x - 1] = image[image_index];
            }
        }

    } else {

        shared_image[threadIdx.x + R] = image[index];
    }

    __syncthreads();


    for (int j = 0; j <= 2*R; j++) {
        int shared_index = threadIdx.x + j;
        shared_results[threadIdx.x] += shared_image[shared_index] * shared_mask[j];

    }
    output[index] = shared_results[threadIdx.x];


}


__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block) {

                        int blockNum = (n + threads_per_block - 1) / threads_per_block;
                        stencil_kernel <<< blockNum, threads_per_block, (2*R+1) * sizeof(float) + (threads_per_block + 2*R) * sizeof(float) + threads_per_block * sizeof(float) >>> (image, mask, output, n, R);
                        hipDeviceSynchronize();
                      }