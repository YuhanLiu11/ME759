#include "hip/hip_runtime.h"
#include "vadd.cuh"

__global__ void vadd(const float *a, float *b, unsigned int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        b[index] = b[index] + a[index];
    }


}