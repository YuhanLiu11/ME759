#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "vadd.cuh"

#define BLOCK_SIZE 512


int main(int argc, char** argv) {
    using namespace std;
    int N = atoi(argv[1]);
    float *a = (float *)malloc(N * sizeof(float));
    float *b = (float *)malloc(N * sizeof(float));
    for(int i = 0; i < N; i++) {
        a[i] = 1;
        b[i] = 1;
    }
    int block = N / BLOCK_SIZE + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float *dA;
    float *dB;
    hipMalloc(&dA, N * sizeof(float));
    hipMalloc(&dB, N * sizeof(float));
    hipMemcpy(dA, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, N*sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);
    vadd <<< block, BLOCK_SIZE >>>(dA, dB, N);

    hipDeviceSynchronize();


    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    float *result = (float *)malloc( N * sizeof(float));
    hipMemcpy(result, dB, N*sizeof(float), hipMemcpyDeviceToHost);

    cout << milliseconds / 1000 << endl;
    cout << result[0] << endl;
    cout << result[N-1] << endl;
    free(result);
    free(b);

    free(a);
    hipFree(&dB);
    hipFree(&dA);

    return 0;
}