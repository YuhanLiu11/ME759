#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>


__global__ void calc(int *dA){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dA[index] = blockIdx.x + threadIdx.x;

}
int main(void) {
    using namespace std;
    int *dA;
    int size = sizeof(int);
    hipMalloc(&dA, 16 * size);
    calc <<< 2 , 8 >>>(dA);
    hipDeviceSynchronize();
    int *hA = (int *)malloc(16 * size);

    hipMemcpy(hA, dA, 16*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 16; i++) {
        if (i < 15)
            cout << hA[i] << " ";
        else {
            cout << hA[i] << endl;
        }
    }

    free(hA);
    hipFree(&dA);
    return 0;
}