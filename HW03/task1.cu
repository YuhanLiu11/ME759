#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>

#include "vadd.cuh"

__global__ void cuda_hello(){
    using namespace std;
    int v = threadIdx.x;
    std::printf("Hello World! I am thread %d.\n", v);
}
int main(int argc, char** argv) {

    cuda_hello <<< 1 , 4 >>>();
    hipDeviceSynchronize();
    return 0;
}