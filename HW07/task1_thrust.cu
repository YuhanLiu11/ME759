#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>



int main(int argc, char** argv) {
    using namespace std;
    using namespace thrust;
    unsigned int n = atol(argv[1]);
    thrust::host_vector<int> h_vec(n);
    thrust::fill(h_vec.begin(), h_vec.end(), 1);


//    cout << "start copying" << endl;
    thrust::device_vector<int> d_vec(n);

    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int tot = thrust::reduce(d_vec.begin(), d_vec.end());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);


    cout << tot << endl;
    cout << ms << endl;


}
