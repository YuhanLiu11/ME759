#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>

using namespace hipcub;

CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char** argv) {
    unsigned int n = atol(argv[1]);
    int *h_in = new int[n];
    for (unsigned int i = 0; i < n; i++) {
        h_in[i] = 1;
    }

    int* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(int) * n));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * n, hipMemcpyHostToDevice));
    int* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(int) * 1));
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    int gpu_sum;
    CubDebugExit(hipMemcpy(&gpu_sum, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));

    std::cout << gpu_sum << std::endl;
    std::cout << ms << std::endl;

    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
}