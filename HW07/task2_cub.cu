#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <hipcub/hipcub.hpp>

#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char** argv) {
    long n = atol(argv[1]);
    float *h_in = new float[n];
    std::fill(h_in, h_in + n, 1);
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));
    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
    float* d_scan = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_scan, sizeof(float) * n));
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_scan, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Do the actual reduce operation
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_scan, n));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    float* gpu_scan = new float[n];
    CubDebugExit(hipMemcpy(gpu_scan, d_scan, sizeof(float) * n, hipMemcpyDeviceToHost));

    std::cout << gpu_scan[n - 1] << endl;
    std::cout << ms << endl;

    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_scan) CubDebugExit(g_allocator.DeviceFree(d_scan));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    free(gpu_scan);
}