#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "count.cuh"
using namespace std;
int main(int argc, char *argv[]) {
    int n = atol(argv[1]);

    thrust::host_vector<int> h_in(n);
    for (int i = 0; i < n; i++) {
        h_in[i] = ((int) rand() % 25);
//        cout << "random number: " << h_in[i] << endl;
    }

    thrust::device_vector<int> d_in = h_in;
    thrust::device_vector<int> values(n);
    thrust::device_vector<int> counts(n);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    count(d_in, values, counts);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
//
//    for (int i = 0; i < int(values.size()); i++) {
//        cout << values[i] << endl;
//        cout << counts[i] << endl;
//    }
    std::cout << values.back() << std::endl;
    std::cout << counts.back() << std::endl;
    std::cout << ms << std::endl;
}