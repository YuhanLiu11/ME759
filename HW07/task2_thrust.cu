#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


int main(int argc, char** argv) {
    using namespace std;
    using namespace thrust;
    long n = atol(argv[1]);
    thrust::host_vector<float> h_vec(n);
    thrust::fill(h_vec.begin(), h_vec.end(), 1);
    thrust::host_vector<float> h_res(n);

//    cout << "start copying" << endl;
    thrust::device_vector<float> d_vec(n);

    thrust::copy(h_vec.begin(), h_vec.end(), d_vec.begin());
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    thrust::device_vector<float> d_intermediate(n);

    thrust::exclusive_scan(d_vec.begin(), d_vec.end(), d_intermediate.begin());



    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    thrust::copy(d_intermediate.begin(), d_intermediate.end(), h_res.begin());
    cout << h_res[n - 1] << endl;
    cout << ms << endl;


}
