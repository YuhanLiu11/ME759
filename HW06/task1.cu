#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "mmul.h"
#include "../include/hipblas.h"

int main(int argc, char *argv[]) {
    using namespace std;
    long n = atol(argv[1]);
    int n_ntests = atol(argv[2]);
    float *A;
    float *B;
    float *C;

    hipMallocManaged(&A, (n * n) * sizeof(float));
    hipMallocManaged(&B, (n * n) * sizeof(float));
    hipMallocManaged(&C, (n * n) * sizeof(float));


    float total_time = 0;
    for (int i = 0; i < n * n; i++) {
        A[i] = 1;
        B[i] = 1;

    }



    for (int time = 0; time < n_ntests; time++) {

        hipEvent_t start;
        hipEvent_t stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        hipblasHandle_t handle ;
        hipblasCreate(&handle);
        mmul(handle, A, B, C, n);
        hipblasDestroy(handle);



        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_time += ms;
        for (int i = 0; i < n * n; i++) {
            C[i] = 0;
        }

    }
    cout << total_time / n_ntests << endl;
    hipFree(A);
    hipFree(B);
    hipFree(C);
}