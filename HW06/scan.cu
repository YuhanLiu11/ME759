#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "scan.cuh"
#include "../include/hip/driver_types.h"

__global__ void scan(float *g_odata, float *g_idata, int n) {
    using namespace std;
    extern volatile __shared__  float temp[]; // allocated on invocation

    int thid = threadIdx.x;
    int pout = 0, pin = 1;
    // load input into shared memory.
    // **exclusive** scan: shift right by one element and set first output to 0
    int global_index = blockIdx.x * blockDim.x + thid;
    if (global_index >= n) {
        temp[thid] = 0;
    } else {
        temp[thid] = g_idata[global_index];
    }

    __syncthreads();




    for( int offset = 1; offset<blockDim.x; offset *= 2 ) {
        pout = 1 - pout; // swap double buffer indices
        pin  = 1 - pout;

        if (thid >= offset)
            temp[pout * blockDim.x + thid] = temp[pin * blockDim.x + thid] + temp[pin * blockDim.x + thid - offset];
        else
            temp[pout*blockDim.x+thid] = temp[pin*blockDim.x+thid];

        __syncthreads(); // I need this here before I start next iteration
    }
    if (pout * blockDim.x + thid < blockDim.x)
        g_odata[global_index] = temp[pout * n + thid];

}

__global__ void copy(float *dOut, float *num_blocks_out, int n, int threads_per_block) {
    int index = (threadIdx.x + 1) * threads_per_block - 1;
    if (index >= n)
        index = n - 1;
//    using namespace std;
//    std::printf("in copy: %f\n", dOut[index]);
    num_blocks_out[threadIdx.x] = dOut[index];
}
__global__ void add(float *num_blocks_out, float *first_output, int n, float *dFinal) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= n)
        return;
    if (blockIdx.x == 0)
        dFinal[index] = first_output[index];
    else
        dFinal[index] = first_output[index] + num_blocks_out[blockIdx.x - 1];

}
__host__ void scan(const float* in, float* out, unsigned int n, unsigned int threads_per_block) {
    using namespace std;

    float *dIn;
    float *dOut;
    hipMalloc(&dIn, n * sizeof(float));
    hipMemcpy(dIn, in, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dOut, n * sizeof(float));
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    scan <<< num_blocks, threads_per_block, 2 * threads_per_block * sizeof(float) >>> (dOut, dIn, n);
    float *hOut = (float *)malloc(n * sizeof(float));
    hipMemcpy(hOut, dOut, n * sizeof(float), hipMemcpyDeviceToHost);

    float *temp_out;
    hipMalloc(&temp_out, num_blocks * sizeof(float));
    copy <<<1, num_blocks>>> (dOut, temp_out, n, threads_per_block);





    float *second_output;
    hipMalloc(&second_output, num_blocks * sizeof(float));
    scan <<< 1, threads_per_block, 2 * threads_per_block * sizeof(float) >>> (second_output, temp_out, num_blocks);


//    float *final_output_host = (float *)malloc(num_blocks * sizeof(float));
//    hipMemcpy(final_output_host, second_output, num_blocks * sizeof(float), hipMemcpyDeviceToHost);
//    for (int i = 0; i < num_blocks; i++) {
//        std::printf("second scan: %f\n", final_output_host[i]);
//    }
//    float *first_output_host = (float *)malloc(n * sizeof(float));
//    hipMemcpy(first_output_host, dOut, n * sizeof(float), hipMemcpyDeviceToHost);
//    for (int i = 0; i < n; i++) {
//        std::printf("first scan: %d %f\n", i, first_output_host[i]);
//    }


    float *dFinal;
    hipMalloc(&dFinal, n * sizeof(float));
    add <<< num_blocks, threads_per_block >>> (second_output, dOut, n, dFinal);
    out[0] = 0;
    hipMemcpy(out + 1, dFinal, (n - 1) * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(dIn);
    hipFree(dOut);
    free(hOut);
    hipFree(temp_out);
    hipFree(second_output);
    hipFree(dFinal);

}