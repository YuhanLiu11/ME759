#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include "scan.cuh"

int main(int argc, char *argv[]) {
    using namespace std;
    long n = atol(argv[1]);
    float *hA;
    float *out;
    hA = (float *)malloc(n * sizeof(float));
    out = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        hA[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;
        out[i] = 0;
    }
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    scan(hA, out, n, 1024);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    cout << out[n-1] << endl;
    cout << ms << endl;
}