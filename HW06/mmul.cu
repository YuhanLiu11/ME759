#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "mmul.h"

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
    int lda = n;
    int ldb = n;
    int ldc = n;
    const float bet = 1;
    const float alf = 1;

    const float *alpha = &alf;
    const float *beta = &bet;
//    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, A, lda, B, ldb, beta, C, ldc);
    hipDeviceSynchronize();

}