#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "reduce.cuh"
__global__ void reduce_kernel(const int* g_idata, int* g_odata, unsigned int n) {
    extern __shared__ int sdata[];
    using namespace std;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        sdata[threadIdx.x] = 0;
    } else {
        sdata[threadIdx.x] = g_idata[index];
    }

    __syncthreads();

    for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
        if(threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) g_odata[blockIdx.x] = sdata[0];

}


__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block) {
    int *g_idata;
    int *g_odata;
    using namespace std;
    hipMalloc(&g_idata, N * sizeof(int));

    hipMemcpy(g_idata, arr, N * sizeof(int), hipMemcpyHostToDevice);


    for (int l = N; l > 1; l = (l + threads_per_block - 1) / threads_per_block) {
        int block_num = (l + threads_per_block - 1) / threads_per_block;
        hipMalloc(&g_odata, block_num * sizeof(int));
        reduce_kernel<<<block_num, threads_per_block, threads_per_block * sizeof(int) >>>(g_idata, g_odata, l);


        hipMemcpy(g_idata, g_odata, block_num * sizeof(int), hipMemcpyDeviceToDevice);

    }

    hipDeviceSynchronize();

    int res;
    hipMemcpy(&res, g_odata, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(g_odata);
    hipFree(g_idata);

    return res;
}