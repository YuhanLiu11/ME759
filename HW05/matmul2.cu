#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "matmul.cuh"

__global__ void
matmul_kernel(const float *A, const float *B, float *C, unsigned int n) {

    int BLOCK_SIZE = blockDim.x;
    int wA = n, wB = n;


    // Shared memory for the sub-matrices (tiles) of  A and B
    extern __shared__ float shared_memory[];
    float *As = shared_memory;
    float *Bs = As + BLOCK_SIZE * BLOCK_SIZE;

    // Block index
    int bx = blockIdx.x; //the B (and C) matrix sub-block column index
    int by = blockIdx.y; //the A (and C) matrix sub-block row index

    // Thread index
    int tx = threadIdx.x; //the column index in the sub-block
    int ty = threadIdx.y; //the row index in the sub-block
    int index =  wB * BLOCK_SIZE * by + BLOCK_SIZE * bx + wB * ty + tx;
    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by; // 0

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + wA - 1; // 4

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE; // 10

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;

    // The element of the block sub-matrix that is computed
    // by the thread
    float Csub = 0;



    // Loop over all the sub-matrices (tiles) of A and B required to
    // compute the block sub-matrix; moving in A left to right in
    // a row, and in B from top to bottom in a column
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep) {

        // Load tiles from global memory into shared memory; each
        // thread loads one element of the two tiles from A & B
        int a_row_index = a / n;
        int a_col_index = a % n;
        int A_index = a + wA * ty + tx;
        int B_index = b + wB * ty + tx;
        int b_row_index = b / n;
        int b_col_index = b % n;
        if (A_index / n >= n || (a_row_index + ty >= wA) || A_index % n < a_col_index
        || (a_col_index + tx >= wA)
        ) {

            As[ty * BLOCK_SIZE + tx] = 0;
        }

        else
          As[ty * BLOCK_SIZE + tx] = A[A_index];
        if (B_index / n >= n || (b_row_index + ty >= wB) || B_index % n < b_col_index
            || (b_col_index + tx >= wB)
                )
            Bs[ty * BLOCK_SIZE + tx] = 0;
        else
            Bs[ty * BLOCK_SIZE + tx] = B[B_index];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();


        // Each thread in this block computes one element
        // of the block sub-matrix (tile).  Thread with indexes
        // ty and tx computes in this tile the entry [ty][tx].

        for (int k = 0; k < BLOCK_SIZE; ++k) {

            Csub += As[ty * BLOCK_SIZE + k] * Bs[k * BLOCK_SIZE + tx];

        }


        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();


    }
    // Write the block sub-matrix to global memory;
    // each thread writes one element

    if (index < n * n && (aBegin % n + threadIdx.x < n) && (aBegin / n + ty < n)
    && (bBegin % n + threadIdx.x < n) && (bBegin / n + threadIdx.y < n)) {
        C[index] = Csub;
    }


}


__host__ void matmul(const float *A, const float *B, float *C, unsigned int n,
                     unsigned int block_dim) {
    using namespace std;
    int blockNum = (n + block_dim - 1) / block_dim;

    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid(blockNum, blockNum);


    matmul_kernel << < dimGrid, dimBlock,
            (block_dim * block_dim) * sizeof(float) +
            (block_dim * block_dim) * sizeof(float) >> > (A, B, C, n);
    hipDeviceSynchronize();
}