#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "reduce.cuh"
int main(int argc, char *argv[]) {
    using namespace std;
    long n = atol(argv[1]);
    long threads_per_block = atol(argv[2]);

    auto arr = new int[n];

    for (int i = 0; i < n; i++) {
        arr[i] = 1;
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    int res = reduce(arr, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    cout << res << endl;
    cout << ms << endl;
}