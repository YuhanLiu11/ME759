#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "matmul.cuh"

__global__ void matmul_kernel(const float* A, const float* B, float* C, unsigned int n) {
    using namespace std;

    extern __shared__ float s[];


    float *shared_A = s;
    float *shared_B = (float *)&shared_A[(blockDim.x * blockDim.y)];
    int bx = blockIdx.x;
    int by = blockIdx.y; // the row index of the block
    int index = n * blockDim.y * by + blockDim.x * bx + n* threadIdx.y + threadIdx.x;
//    std::printf("index: %f\n", A[n*n-1]);

    if (index >= n*n)
        return;

    int aBegin = n * blockDim.x * by; // (by * blockDim.x, 0) index in A
    int bBegin = blockDim.y * bx;  //
    int aEnd = n + aBegin - 1;
    int aStep = blockDim.x;
    int bStep = blockDim.y * n;

    float Csub = 0; // The output that the thread calculates
    for (size_t i = aBegin, j = bBegin; i <= aEnd; i += aStep, j += bStep) {
        std::printf("added %d\n",  threadIdx.y * n + threadIdx.x + i);
        shared_A[threadIdx.y * n + threadIdx.x] = A[threadIdx.y * n + threadIdx.x + i];
        shared_B[threadIdx.y * n + threadIdx.x] = B[threadIdx.y * n + threadIdx.x + j];

        __syncthreads();

        // all threads in this block have done copying data


    }
    std::printf("added \n");
    for (size_t k = 0; k < blockDim.x; k++) {
        Csub += shared_A[threadIdx.y * n + k] * shared_B[k * n + threadIdx.x];
        std::printf("added \n");

    }
    __syncthreads();
    C[index] = Csub;


}


__host__ void matmul(const float* A, const float* B, float* C, unsigned int n, unsigned int block_dim) {
    using namespace std;
    int blockNum = (n + block_dim - 1) / block_dim;

    dim3 dimBlock (block_dim, block_dim);
    dim3 dimGrid (blockNum, blockNum);


    matmul_kernel<<< dimGrid, dimBlock, (block_dim * block_dim) * sizeof(float) + (block_dim * block_dim) * sizeof(float) >>>(A, B, C, n);
    hipDeviceSynchronize();
}