#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "matmul.cuh"

int main(int argc, char** argv) {
    using namespace std;

    long n = atol(argv[1]);
    int block_dim = atoi(argv[2]);
    float *A;
    float *B;
    float *C;
    hipMallocManaged(&A, (n * n) * sizeof(float));
    hipMallocManaged(&B, (n * n) * sizeof(float));
    hipMallocManaged(&C, (n * n) * sizeof(float));
    for (int i = 0; i < n * n; i++) {
        A[i] = 1;
        B[i] = 1;
        // initialize C
        C[i] = 0;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    unsigned int N = (unsigned int) n;
    matmul(A, B, C, N, block_dim);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << C[0] << endl;
    cout << C[n * n - 1] << endl;
    cout << milliseconds << endl;





    hipFree(A);
    hipFree(B);
    hipFree(C);

}